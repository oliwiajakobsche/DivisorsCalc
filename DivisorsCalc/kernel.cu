#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <iostream>


using namespace std;

__global__ void InsertIntoCTabIsDivisible(int* a, int number, int nrOfThreads) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nrOfThreads)
    {
        if ((number % (threadId + 1) == 0))
            a[threadId] = 1;
        else
            a[threadId] = 0;
    }
}

__global__ void InsertIntoETabDividors(int* c, int* d, int* e, int nrOfThreads) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nrOfThreads)
    {
        if (c[threadId] == 1)
        {
            int temp = d[threadId];
            e[temp] = threadId + 1;
        }
    }
}

int main()
{
    PrintProjectInfo();
    int y = GetNumberFromUser();
    int n = MaxDividorToCheck(y);

    //************** PREPARING MEMORY **************
    int blockSize = 1024;
    dim3 threadsAmmount(blockSize);
    dim3 blocksAmmount((n / blockSize) + 1);
    //Alokacja pamięci na tablice po stronie CPU (host)
    int* c = new int[n];
    int* d = new int[n];
    int* e = new int[n];
    //Alokacja pamięci na tablice po stronie GPU (device)
    int* device_c;
    int* device_d;
    int* device_e;
    const int size = n * sizeof(int);
    hipMalloc((void**)&device_c, size);
    hipMalloc((void**)&device_d, size);
    hipMalloc((void**)&device_e, size);
    //**********************************************

                                                                                                 
    InsertIntoCTabIsDivisible<<<blocksAmmount, blockSize>>>(device_c, y, n);    
    hipMemcpy(c, device_c, size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < n; i++)
    {
        cout << "c[" << i << "] =" << c[i] << endl;
    }

    exclusive_scan(thrust::device, device_c, device_c + n, device_d, 0);
    hipMemcpy(d, device_d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        cout << "d[" << i << "] =" << d[i] << endl;
    }

    InsertIntoETabDividors<<<blocksAmmount, blockSize>>>(device_c, device_d, device_e, n);

    hipMemcpy(e, device_e, size, hipMemcpyDeviceToHost);

    int dividorsCounter = c[n - 1] + d[n - 1];

    for (int i = 0; i < dividorsCounter; i++)
    {
        cout << e[i] <<endl;
    }

    return 0;
}

int MaxDividorToCheck(int y)
{
    int dividorsNr = floor(sqrt(y));
    cout << "Max possible dividor to check - floor(sqrt(y)) = " << dividorsNr <<endl;
    return dividorsNr;
}

void PrintProjectInfo()
{
    cout << "*******************" << endl;
    cout << "Divisors Generator" << endl;
    cout << "Student nr 106545" << endl;
    cout << "*******************\n" << endl;
}

int GetNumberFromUser()
{
    int input = 0;

    do {
        cout << "Enter an integer value to get the list of divisors of this number: ";
        cin >> input;
    } while (input < 1);

    return input;
}

