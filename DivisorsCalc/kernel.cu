#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
#include <iostream>

using namespace std;

int main()
{
    PrintProjectInfo();
    int y = GetNumberFromUser();
    int n = MaxDividorToCheck(y);

    // ************** PREPARING MEMORY **************
    int blockSize = 1024;
    dim3 threadsAmmount(blockSize);
    dim3 blocksAmmount((n / blockSize) + 1);
    blocksAmmount.x = ((n) / blockSize) + 1;
    //Alokacja pamięci na tablice po stronie CPU (host)
    int* c = new int[n];
    int* d = new int[n];
    int* e = new int[n];
    //Alokacja pamięci na tablice po stronie GPU (device)
    int* device_c;
    int* device_d;
    int* device_e;
    const int size = n * sizeof(int);
    hipMalloc((void**)&device_c, size);
    hipMalloc((void**)&device_d, size);
    hipMalloc((void**)&device_e, size);
    // **********************************************

    return 0;
}

int MaxDividorToCheck(int y)
{
    int dividorsNr = floor(sqrt(y));
    cout << "Max possible dividor to check - floor(sqrt(y)) = " << dividorsNr;
    return dividorsNr;
}

void PrintProjectInfo()
{
    cout << "*******************" << endl;
    cout << "Divisors Generator" << endl;
    cout << "Student nr 106545" << endl;
    cout << "*******************\n" << endl;
}

int GetNumberFromUser()
{
    int input = 0;

    do {
        cout << "Enter an integer value to get the list of divisors of this number: ";
        cin >> input;
    } while (input < 1);

    return input;
}

