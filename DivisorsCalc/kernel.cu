#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <iostream>

using namespace std;

__global__ void InsertIntoCTabIsDivisible(int* a, int number, int nrOfThreads) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nrOfThreads)
    {
        if ((number % (threadId + 1) == 0))
            a[threadId] = 1;
        else
            a[threadId] = 0;
    }
}

__global__ void InsertIntoETabDividors(int* c, int* d, int* e, int nrOfThreads) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nrOfThreads)
    {
        if (c[threadId] == 1)
        {
            int temp = d[threadId];
            e[temp] = threadId + 1;
        }
    }
}

int main()
{
    PrintProjectInfo();
    int y = GetNumberFromUser();

    hipError_t cudaStatus = hipSetDevice(0);;
    const clock_t begin_time = clock();
    int n = MaxDividorToCheck(y);

    //************** PREPARING MEMORY **************
    int blockSize = 1024;
    dim3 threadsAmmount(blockSize);
    dim3 blocksAmmount((n / blockSize) + 1);
    //Memory allocation for tab on CPU side (host)
    int* c = new int[n];
    int* d = new int[n];
    int* e = new int[n];
    //Memory allocation for tab on GPU side (device)
    int* device_c;
    int* device_d;
    int* device_e;
    const int size = n * sizeof(int);
    hipMalloc((void**)&device_c, size);
    hipMalloc((void**)&device_d, size);
    hipMalloc((void**)&device_e, size);
    //**********************************************
                                                                                                     
    InsertIntoCTabIsDivisible<<<blocksAmmount, blockSize>>>(device_c, y, n);   
    hipDeviceSynchronize();
    hipMemcpy(c, device_c, size, hipMemcpyDeviceToHost);    
    PrintTab("c", c, n);

    exclusive_scan(thrust::device, device_c, device_c + n, device_d, 0);
    hipMemcpy(d, device_d, size, hipMemcpyDeviceToHost);
    PrintTab("d", d, n);

    InsertIntoETabDividors<<<blocksAmmount, blockSize>>>(device_c, device_d, device_e, n);
    hipDeviceSynchronize();
    hipMemcpy(e, device_e, size, hipMemcpyDeviceToHost);
    int dividorsCounter = c[n - 1] + d[n - 1];
    PrintTab("e", e, dividorsCounter);
    cout << "Time: " << float(clock() - begin_time) / CLOCKS_PER_SEC << " s " << endl;

    //Release memory
    delete[] c; delete[] d; delete[] e;    
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return EXIT_SUCCESS;
}

void PrintTab(char* tabName, int* tab, int iterateTo)
{
    for (int i = 0; i < iterateTo; i++)
    {
        cout << tabName << "[" << i << "] = " << tab[i] << endl;
    }

    cout << endl;
}

int MaxDividorToCheck(int y)
{
    int dividorsNr = floor(sqrt(y));
    cout << "Max possible dividor to check - floor(sqrt(y)) = " << dividorsNr <<endl;
    return dividorsNr;
}

void PrintProjectInfo()
{
    cout << "*******************" << endl;
    cout << "Divisors Generator" << endl;
    cout << "Student nr 106545" << endl;
    cout << "*******************\n" << endl;
}

int GetNumberFromUser()
{
    int input = 0;

    do {
        cout << "Enter an integer value to get the list of divisors of this number: ";
        cin >> input;
    } while (input < 1);

    return input;
}

