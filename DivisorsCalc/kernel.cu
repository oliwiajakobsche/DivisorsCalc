#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include <stdio.h>
#include <iostream>

using namespace std;

int main()
{
    PrintProjectInfo();
    int y = GetNumberFromUser();
    int n = DetermineNumberOfDivisorsToCheck(y);

   
    return 0;
}

int DetermineNumberOfDivisorsToCheck(int y)
{
    int dividorsNr = floor(sqrt(y));
    cout << "Max possible dividor to check - floor(sqrt(y)) = " << dividorsNr;
    return dividorsNr;
}

void PrintProjectInfo()
{
    cout << "*******************" << endl;
    cout << "Divisors Generator" << endl;
    cout << "Student nr 106545" << endl;
    cout << "*******************\n" << endl;
}

int GetNumberFromUser()
{
    int input = 0;

    do {
        cout << "Enter an integer value to get the list of divisors of this number: ";
        cin >> input;
    } while (input < 1);

    return input;
}

